
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<time.h>
//#include<cuda_runtime.h>
#define N 1000000

__global__ void is_prime(int* d_count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= 3 && idx < N)
    {
        int flag = 0;
        for(int j=2; j<idx && flag == 0; j++)
        {
            if((idx % j) == 0)
            {
                flag = 1;
            }
        }
        if(flag == 0)
        {
            atomicAdd(d_count, 1);
        }
    }
}
 int main()
{
    int count = 1;
    int* d_count;
    hipMalloc(&d_count, sizeof(int));
    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    clock_t start_time = clock();
    is_prime<<<blocksPerGrid, threadsPerBlock>>>(d_count);
    
    hipDeviceSynchronize();
    clock_t stop_time = clock();
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_count);
    
    double exe_time = ((double) (stop_time - start_time)) / CLOCKS_PER_SEC;
    printf("\n Number of prime numbers = %d \n Execution time is = %lf seconds\n", count, exe_time);
    return 0;
}
